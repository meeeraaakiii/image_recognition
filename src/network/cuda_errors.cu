#include "hip/hip_runtime.h"
// cuda_errors.cu
#include <hip/hip_runtime.h>
#include <execinfo.h>  // For backtrace

#include <cstdlib>
#include <iostream>

#include "io/colors.h"
#include "io/log.h"

inline void printBacktrace() {
    const int maxFrames = 100;
    void* frames[maxFrames];
    int frameCount = backtrace(frames, maxFrames);
    char** symbols = backtrace_symbols(frames, frameCount);

    LOG(SL::ERROR) << GREY << "Backtrace:" << std::endl;
    for (int i = 0; i < frameCount; ++i) {
        LOG(SL::ERROR) << symbols[i] << std::endl;
    }
    LOG(SL::ERROR) << RESET;

    free(symbols);
}

inline void checkCudaError(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        LOG(SL::ERROR) << RED << "\nCUDA Error: " << hipGetErrorString(error) << " at " << file << ":" << line << RESET
                       << std::endl;
        printBacktrace();
        std::exit(EXIT_FAILURE);
    }
}

// Macro to simplify usage of the inline function
#define CUDA_CHECK_ERROR(call) checkCudaError((call), __FILE__, __LINE__)

// Wrapper for hipMalloc with the same signature
void cudaMallocWrapper(void** ptr, size_t size) { CUDA_CHECK_ERROR(hipMalloc(ptr, size)); }

// Wrapper for hipMemcpy
void cudaMemcpyWrapper(void* dst, const void* src, size_t size, hipMemcpyKind kind) {
    CUDA_CHECK_ERROR(hipMemcpy(dst, src, size, kind));
}

// Wrapper for hipFree
void cudaFreeWrapper(void* ptr) { CUDA_CHECK_ERROR(hipFree(ptr)); }
